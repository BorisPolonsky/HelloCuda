
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <sys/time.h>

#define FALSE 0
#define TRUE !FALSE

double cpuSecond()
{
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

__global__ void warmUp(double *arr, const int arrSize)
{
	//The same as minimalWarpDivergence()
	int i=threadIdx.x+threadIdx.y*blockDim.x+threadIdx.z*blockDim.x*blockDim.y;
	if(i<arrSize)
	{	
		if(TRUE)
			arr[i]=i%2;
		else
			arr[i]=i%2;
	}
}


__global__ void simpleWarpDivergence(double *arr, const int arrSize)
{
	int i=threadIdx.x+threadIdx.y*blockDim.x+threadIdx.z*blockDim.x*blockDim.y;
	if(i<arrSize)
	{	
		if(i%2==0)
			arr[i]=0;
		else
			arr[i]=1;
	}
}

__global__ void minimalWarpDivergence(double *arr, const int arrSize)
{
	int i=threadIdx.x+threadIdx.y*blockDim.x+threadIdx.z*blockDim.x*blockDim.y;
	if(i<arrSize)
	{	
		if(TRUE)
			arr[i]=i%2;
		else
			arr[i]=i%2;
	}
}

__global__ void exposedWarpDivergence(double *arr, const int arrSize)
{
	int i=threadIdx.x+threadIdx.y*blockDim.x+threadIdx.z*blockDim.x*blockDim.y;
	if(i<arrSize)
	{	
		bool flag=(i%2==0);
		if(flag)
			arr[i]=0;
		else
			arr[i]=1;
	}
}

int main()
{
	const int arrSize=1<<16;
	double t=0;
	double *d_arr=NULL;
	if(hipMalloc((void**)&d_arr, arrSize*sizeof(double))!=hipSuccess)
	{
		printf("Failed to allocated enough memory on GPU.\n");
		exit(-1);
	}
	dim3 grid(1<<8), block(1<<8);

	hipMemset(d_arr, 0, arrSize*sizeof(double));
	t=cpuSecond();
	warmUp<<<grid, block>>>(d_arr,arrSize);
	hipDeviceSynchronize();
	t=cpuSecond()-t;
	printf("Warm up took %lf s.\n", t);

	hipMemset(d_arr, 0, arrSize*sizeof(double));
	t=cpuSecond();
	simpleWarpDivergence<<<grid, block>>>(d_arr,arrSize);
	hipDeviceSynchronize();
	t=cpuSecond()-t;
	printf("Array initialization with simple warp divergence took %lf s.\n", t);

	hipMemset(d_arr, 0, arrSize*sizeof(double));
	t=cpuSecond();
	minimalWarpDivergence<<<grid, block>>>(d_arr,arrSize);
	hipDeviceSynchronize();
	t=cpuSecond()-t;
	printf("Array initialization with minimal warp divergence took %lf s.\n", t);

	hipMemset(d_arr, 0, arrSize*sizeof(double));
	t=cpuSecond();
	exposedWarpDivergence<<<grid, block>>>(d_arr,arrSize);
	hipDeviceSynchronize();
	t=cpuSecond()-t;
	printf("Array initialization with exposed warp divergence took %lf s.\n", t);
	hipFree(d_arr);
	hipDeviceReset();
	return 0;
}
