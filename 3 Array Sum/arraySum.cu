
#include <hip/hip_runtime.h>
#include <stdio.h>

void setArray(float *, int);
void cpuSum(float *, float *, float *, int);
void checkCpuMalloc(float *);
void checkGpuMalloc(hipError_t);
int arrEqual(float*, float*, int);

int main()
{
	int num = 64;
	float *h_x1 =(float *)malloc(num * sizeof(float));
	float *h_x2 =(float *)malloc(num * sizeof(float));
	float *h_y1 =(float *)malloc(num * sizeof(float));
	float *h_y2 =(float *)malloc(num * sizeof(float));

	checkCpuMalloc(h_x1);
	checkCpuMalloc(h_x2);
	checkCpuMalloc(h_y1);
	checkCpuMalloc(h_y2);

	setArray(h_x1, num);
	setArray(h_x2, num);
	
	float *d_x1 = NULL, *d_x2 = NULL, *d_y = NULL;
	checkGpuMalloc(hipMalloc((void**)&d_x1, num * sizeof(float)));
	checkGpuMalloc(hipMalloc((void**)&d_x2, num * sizeof(float)));
	checkGpuMalloc(hipMalloc((void**)&d_y, num * sizeof(float)));
	hipMemcpy(d_x1, h_x1, num * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x2, h_x2, num * sizeof(float), hipMemcpyHostToDevice);
	cpuSum(h_x1, h_x2, h_y1, num);
	//To be implemented
	
	hipMemcpy(h_y2, d_y, num * sizeof(float), hipMemcpyDeviceToHost);
	if(arrEqual(h_y1, h_y2, num))
		printf("Equal\n");
	else
		printf("Not equal.\n");
	free(h_x1);
	free(h_x2);
	free(h_y1);
	free(h_y2);
	hipFree(d_x1);
	hipFree(d_x2);
	hipFree(d_y);
	hipDeviceReset();
	return 0;
}

void setArray(float *p, int num)
{
	for(int i = 0; i < num; i++)
	{
		p[i] = 0.618 * i;	
	}
}

void cpuSum(float *x1, float *x2, float *y, int num)
{
	for(int i = 0; i < num; i++)
	{
		y[i] = x1[i] +x2[i];
	}
}

int arrEqual(float *arr1, float *arr2, int num)
{
	for(int i=0; i < num; i++)
	{
		if(arr1[i] != arr2[i])
			return 0;
	}
	return 1;
}

void checkCpuMalloc(float *p)
{
	if(p == NULL)
		exit(-1);
}

void checkGpuMalloc(hipError_t code)
{
	if(code != hipSuccess)
		exit(-1);
}
