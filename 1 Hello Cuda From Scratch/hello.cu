
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void)
{
	printf("Hello Cuda 8.0!(From GPU thread no: %d)\n",threadIdx.x);
}

int main()
{
	printf("Hello Cuda 8.0!(From CPU)\n");
	helloFromGPU<<<1, 10>>>();
	hipDeviceReset();
	return 0;
}


